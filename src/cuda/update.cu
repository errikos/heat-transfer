#include "hip/hip_runtime.h"
#include <lcutil.h>
#include <timestamp.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <string.h>

#define BLOCK_SIZE 16  // The conservative approach


__global__ void updateKernel(unsigned int height, unsigned int width,
                             double *wgrid, double *ogrid) {
  // Determine coordinates of block within the data array and launch updates
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (!(i < height) || i < 0 || !(j < width) || j < 0) return;

  double old_val = *(ogrid + i*width + j);
  double left = j ? *(ogrid + i*width + (j-1)) : 0.0f;
  double top = i ? *(ogrid + (i-1)*width + j) : 0.0f;
  double right = (j != width-1) ? *(ogrid + i*width + (j+1)) : 0.0f;
  double bottom = (i != height-1) ? *(ogrid + (i+1)*width + j) : 0.0f;

  *(wgrid + i*width + j) = old_val
                           + 0.1 * (top + bottom - 2.0 * old_val)
                           + 0.1 * (right + left - 2.0 * old_val);
}


extern "C"
int updateGPU(double *host_array, unsigned int height, unsigned int width,
              unsigned int steps, float *elapsed_time) {

  unsigned int block_size = height * width;
  double *grids[2];

  // Allocate space for the the two grids in the GPU
  for (unsigned int i = 0; i != 2; ++i)
    CUDA_SAFE_CALL(hipMalloc(&grids[i], block_size * sizeof(double)));

  // grids[0] will hold the initial data
  CUDA_SAFE_CALL(hipMemcpy(grids[0], host_array, block_size * sizeof(double),
                            hipMemcpyHostToDevice));
  // grids[1] initialized to zeroes
  CUDA_SAFE_CALL(hipMemset(grids[1], 0, block_size * sizeof(double)));

  // Determine grid dimensions according to input array
  // x = ceil(height / BLOCK_SIZE)
  // y = ceil(width / BLOCK_SIZE)
  dim3 blocks(FRACTION_CEILING(height, BLOCK_SIZE),
              FRACTION_CEILING(width, BLOCK_SIZE));
  // BLOCK_SIZE x BLOCK_SIZE threads per block
  dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

  int wgrid = 1;

  timestamp t_start;
  t_start = getTimestamp();  // Start timer

  for (unsigned int i = 0; i != steps; ++i) {
    // Fire update in kernel
    updateKernel<<<blocks, threads>>>(height, width,
                                      grids[wgrid], grids[1-wgrid]);
    // Wait for threads to reach this point
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    wgrid = 1 - wgrid;
  }

  *elapsed_time = getElapsedTime(t_start) / 1000.0f;  // End timer

  // Get results back
  CUDA_SAFE_CALL(hipMemcpy(host_array, grids[1-wgrid],
                            block_size * sizeof(double),
                            hipMemcpyDeviceToHost));

  // Free space allocated in the GPU
  for (unsigned int i = 0; i != 2; ++i)
    CUDA_SAFE_CALL(hipFree(grids[i]));

  return 0;
}
